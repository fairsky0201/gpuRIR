#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>

#include <vector>
#include "gpuRIR_cuda.h"

#if CUDART_VERSION < 9000
#define __h2div h2div
#endif

// Image Source Method
static const int nThreadsISM_x = 4;
static const int nThreadsISM_y = 4;
static const int nThreadsISM_z = 4;

// RIR computation
static const int initialReductionMin = 512;
static const int lut_oversamp = 16; 
static const int nThreadsGen_t = 32;
static const int nThreadsGen_m = 4;
static const int nThreadsGen_n = 1; // Don't change it
static const int nThreadsRed = 128;

// Power envelope prediction
static const int nThreadsEnvPred_x = 4;
static const int nThreadsEnvPred_y = 4;
static const int nThreadsEnvPred_z = 1; // Don't change it

// Generate diffuse reverberation
static const int nThreadsDiff_t = 16;
static const int nThreadsDiff_src = 4;
static const int nThreadsDiff_rcv = 2;

// RIR filtering onvolution
static const int nThreadsConv_x = 256;
static const int nThreadsConv_y = 1;
static const int nThreadsConv_z = 1;

#if __CUDA_ARCH__ >= 530
#define h2zeros __float2half2_rn(0.0)
#define h2ones __float2half2_rn(1.0)
#define h2pi __float2half2_rn(PI)
#endif

// To hide the cuRAND generator in the header and don't need to include the cuda headers there
struct cuRandGeneratorWrapper_t
{
   hiprandGenerator_t gen;
};
cuRandGeneratorWrapper_t gpuRIR_cuda::cuRandGenWrap;

// CUDA architecture in format xy0
int cuda_arch;


/***************************/
/* Auxiliar host functions */
/***************************/

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hiprandStatus_t code, const char *file, int line, bool abort=true) {
   if (code != HIPRAND_STATUS_SUCCESS) 
   {
      fprintf(stderr,"cuRAND: %d %s %d\n", code, file, line);
      if (abort) exit(code);
   }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipfftResult_t code, const char *file, int line, bool abort=true) {
   if (code != HIPFFT_SUCCESS) 
   {
      fprintf(stderr,"cuFFT error: %d %s %d\n", code, file, line);
      if (abort) exit(code);
   }
}

inline unsigned int pow2roundup (unsigned int x) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return x+1;
}

/*****************************/
/* Auxiliar device functions */
/*****************************/

__device__ __forceinline__ scalar_t hanning_window(scalar_t t, scalar_t Tw) {
	return 0.5f * (1.0f + __cosf(2.0f*PI*t/Tw));
}

__device__ __forceinline__ scalar_t sinc(scalar_t x) {
	return (x==0)? 1 : sinf(x)/x;
}

__device__ __forceinline__ scalar_t image_sample(scalar_t amp, scalar_t tau, scalar_t t, int Tw_2, hipTextureObject_t sinc_lut, float lut_center) {
	scalar_t t_tau = t - tau;
	return (abs(t_tau)<Tw_2)? amp * tex1D<scalar_t>(sinc_lut, __fmaf_rz(t_tau,lut_oversamp,lut_center)) : 0.0f;
}

__device__ __forceinline__ scalar_t SabineT60( scalar_t room_sz_x, scalar_t room_sz_y, scalar_t room_sz_z,
							scalar_t beta_x1, scalar_t beta_x2, scalar_t beta_y1, scalar_t beta_y2, scalar_t beta_z1, scalar_t beta_z2 ) {
	scalar_t Sa = ((1.0f-beta_x1*beta_x1) + (1.0f-beta_x2*beta_x2)) * room_sz_y * room_sz_z +
				  ((1.0f-beta_y1*beta_y1) + (1.0f-beta_y2*beta_y2)) * room_sz_x * room_sz_z +
				  ((1.0f-beta_z1*beta_z1) + (1.0f-beta_z2*beta_z2)) * room_sz_x * room_sz_y;
	scalar_t V = room_sz_x * room_sz_y * room_sz_z;
	return 0.161f * V / Sa;
}

__device__ __forceinline__ scalar_t mic_directivity(scalar_t doaVec[3], scalar_t orVec[3], micPattern pattern) {
	if (pattern == DIR_OMNI) return 1.0f;
	
	scalar_t cosTheta = doaVec[0]*orVec[0] + doaVec[1]*orVec[1] + doaVec[2]*orVec[2];
	cosTheta /= sqrtf(doaVec[0]*doaVec[0] + doaVec[1]*doaVec[1] + doaVec[2]*doaVec[2]);
	cosTheta /= sqrtf(orVec[0]*orVec[0] + orVec[1]*orVec[1] + orVec[2]*orVec[2]);
	
	switch(pattern) {
		case DIR_HOMNI:		return (cosTheta>0.0f)? 1.0f : 0.0f;
		case DIR_CARD: 		return 0.5f  +  0.5f*cosTheta;
		case DIR_HYPCARD:	return 0.25f + 0.75f*cosTheta;
		case DIR_SUBCARD: 	return 0.75f + 0.25f*cosTheta;
		case DIR_BIDIR: 	return cosTheta;
		default: printf("Invalid microphone pattern"); return 0.0f;
	}
}

// hipfftComplex scale
__device__ __forceinline__ hipfftComplex ComplexScale(hipfftComplex a, float s) {
    hipfftComplex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// hipfftComplex multiplication
__device__ __forceinline__ hipfftComplex ComplexMul(hipfftComplex a, hipfftComplex b) {
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

/*********************************************/
/* Mixed precision auxiliar device functions */
/*********************************************/
#if __CUDA_ARCH__ >= 530

__device__ __forceinline__ half2 h2abs(half2 x) {
	uint32_t i = *reinterpret_cast<uint32_t*>(&x) & 0x7FFF7FFF;
	return *reinterpret_cast<half2*>( &i );
}

__device__ __forceinline__ half2 my_h2sinpi(half2 x) {
	// Argument reduction to [-0.5, 0.5]
	half2 i = h2rint(x);
	half2 r = __hsub2(x, i);
	
	// sin(pi*x) polinomial approximation for x in [-0.5,0.5]
	half2 r2 = __hmul2(r, r);
	half2 s = __float2half2_rn(+2.31786431325108f);
	s = __hfma2(r2, s, __float2half2_rn(-5.14167814230801f));
	s = __hfma2(r2, s, __float2half2_rn(+3.14087446786993f));
	s = __hmul2(s, r);
	
	half2 i_2 = __hmul2(i, __float2half2_rn(0.5f));
	half2 sgn = __hfma2(__float2half2_rn(-2.0f), 
						__hne2(__hsub2(i_2, h2rint(i_2)), h2zeros), 
						h2ones); // 1 if i is even, else -1: -2 * ((i/2-round(i/2))!=0) + 1
	s = __hmul2(s, sgn);
	
	return s;
}

__device__ __forceinline__ half2 my_h2cospi(half2 x) {
	// It is always on [-0.5, 0.5], so we do not need argument reduction
	
	// cos(pi*x) polinomial approximation for x in [-0.5,0.5]
	half2 x2 = __hmul2(x, x);
	half2 c = __float2half2_rn(-1.229339658587166f);
	c = __hfma2(x2, c, __float2half2_rn(+4.043619929856572f));
	c = __hfma2(x2, c, __float2half2_rn(-4.934120365987677f));
	c = __hfma2(x2, c, __float2half2_rn(+0.999995282317910f));
	
	return c;
}

__device__ __forceinline__ half2 hanning_window_mp(half2 t, half2 Tw_inv) {	
	half2 c = my_h2cospi(__hmul2(Tw_inv, t));
	return __hmul2(c, c);
}

__device__ __forceinline__ half2 my_h2sinc(half2 x) {
	x = __hfma2(__heq2(x, h2zeros), __float2half2_rn(1e-7f), x);
	return __h2div(my_h2sinpi(x), __hmul2(h2pi, x));
	
}

__device__ __forceinline__ half2 image_sample_mp(half2 amp, scalar_t tau, scalar_t t1, scalar_t t2, scalar_t Tw_2, half2 Tw_inv) {
	scalar_t t1_tau = t1-tau;
	scalar_t t2_tau = t2-tau;
	half2 t_tau = __floats2half2_rn(t1_tau, t2_tau);
	if (abs(t1_tau)<Tw_2 || abs(t2_tau)<Tw_2) { // __hble2() is terribly slow
		return __hmul2(hanning_window_mp(t_tau, Tw_inv), __hmul2(amp, my_h2sinc( t_tau )));
	} else return h2zeros;
}

#endif

/***********/
/* KERNELS */
/***********/

__global__ void calcAmpTau_kernel(scalar_t* g_amp /*[M_src]M_rcv][nb_img_x][nb_img_y][nb_img_z]*/, 
								  scalar_t* g_tau /*[M_src]M_rcv][nb_img_x][nb_img_y][nb_img_z]*/, 
								  scalar_t* g_tau_dp /*[M_src]M_rcv]*/,
								  scalar_t* g_pos_src/*[M_src][3]*/, scalar_t* g_pos_rcv/*[M_rcv][3]*/, scalar_t* g_orV_rcv/*[M_rcv][3]*/,
								  micPattern mic_pattern, scalar_t room_sz_x, scalar_t room_sz_y, scalar_t room_sz_z,
								  scalar_t beta_x1, scalar_t beta_x2, scalar_t beta_y1, scalar_t beta_y2, scalar_t beta_z1, scalar_t beta_z2, 
								  int nb_img_x, int nb_img_y, int nb_img_z,
								  int M_src, int M_rcv, scalar_t c, scalar_t Fs) {
	
	extern __shared__ scalar_t sdata[];
		
	int n[3];
	n[0] = blockIdx.x * blockDim.x + threadIdx.x;
	n[1] = blockIdx.y * blockDim.y + threadIdx.y;
	n[2] = blockIdx.z * blockDim.z + threadIdx.z;
	
	int N[3];
	N[0] = nb_img_x;
	N[1] = nb_img_y;
	N[2] = nb_img_z;
	
	scalar_t room_sz[3];
	room_sz[0] = room_sz_x;
	room_sz[1] = room_sz_y;
	room_sz[2] = room_sz_z;
	
	scalar_t beta[6];
	beta[0] = - beta_x1;
	beta[1] = - beta_x2;
	beta[2] = - beta_y1;
	beta[3] = - beta_y2;
	beta[4] = - beta_z1;
	beta[5] = - beta_z2;
	
	int prodN = N[0]*N[1]*N[2];
	int n_idx = n[0]*N[1]*N[2] + n[1]*N[2] + n[2];
	
	// Copy g_pos_src to shared memory
	scalar_t* sh_pos_src = (scalar_t*) sdata;
	if (threadIdx.y==0 && threadIdx.z==0)  {
		for (int m=threadIdx.x; m<M_src; m+=blockDim.x) {
			sh_pos_src[m*3  ] = g_pos_src[m*3  ];
			sh_pos_src[m*3+1] = g_pos_src[m*3+1];
			sh_pos_src[m*3+2] = g_pos_src[m*3+2];
		}
	}
	
	// Copy g_pos_rcv to shared memory
	scalar_t* sh_pos_rcv = &sh_pos_src[M_src*3];
	if (threadIdx.x==0 && threadIdx.z==0)  {
		for (int m=threadIdx.y; m<M_rcv; m+=blockDim.y) {
			sh_pos_rcv[m*3  ] = g_pos_rcv[m*3  ];
			sh_pos_rcv[m*3+1] = g_pos_rcv[m*3+1];
			sh_pos_rcv[m*3+2] = g_pos_rcv[m*3+2];
		}
	}
	
	// Copy g_orV_rcv to shared memory
	scalar_t* sh_orV_rcv = &sh_pos_rcv[M_rcv*3];
	if (threadIdx.x==0 && threadIdx.y==0)  {
		for (int m=threadIdx.z; m<M_rcv; m+=blockDim.z) {
			sh_orV_rcv[m*3  ] = g_orV_rcv[m*3  ];
			sh_orV_rcv[m*3+1] = g_orV_rcv[m*3+1];
			sh_orV_rcv[m*3+2] = g_orV_rcv[m*3+2];
		}
	}
	
	// Wait until the copies are completed
	__syncthreads();
	
	if (n[0]<N[0] & n[1]<N[1] & n[2]<N[2]) {
		
		// Common factors for each src and rcv
		scalar_t rflx_att = 1;
		scalar_t clust_pos[3];
		int clust_idx[3];
		int rflx_idx[3];
		bool direct_path = true;
		for (int d=0; d<3; d++) {
			clust_idx[d] = __float2int_ru((n[d] - N[d]/2) / 2.0f); 
			clust_pos[d] = clust_idx[d] * 2*room_sz[d];
			rflx_idx[d] = abs((n[d] - N[d]/2) % 2); // 1 means reflected in dimension d
			rflx_att *= powf(beta[d*2], abs(clust_idx[d]-rflx_idx[d])) * powf(beta[d*2+1], abs(clust_idx[d]));
			direct_path *= (clust_idx[d]==0)&&(rflx_idx[d]==0);
		}
			
		// Individual factors for each src and rcv
		for (int m_src=0; m_src<M_src; m_src++) {
			for (int m_rcv=0; m_rcv<M_rcv; m_rcv++) {
				scalar_t vec[3];
				scalar_t dist = 0;
				for (int d=0; d<3; d++) {
					vec[d] = clust_pos[d] + (1-2*rflx_idx[d]) * sh_pos_src[m_src*3+d] - sh_pos_rcv[m_rcv*3+d];
					dist += vec[d] * vec[d];
				}
				dist = sqrtf(dist);
				scalar_t amp = rflx_att / (4*PI*dist);
				amp *= mic_directivity(vec, &sh_orV_rcv[m_rcv], mic_pattern);
				g_amp[m_src*M_rcv*prodN + m_rcv*prodN + n_idx] = amp;
				g_tau[m_src*M_rcv*prodN + m_rcv*prodN + n_idx] = dist / c * Fs;

				if (direct_path) g_tau_dp[m_src*M_rcv + m_rcv] = dist / c * Fs;
			}
		}
	}
}

__global__ void generateRIR_kernel(scalar_t* initialRIR, scalar_t* amp, scalar_t* tau, int T, int M, int N, int iniRIR_N, int ini_red, int Tw_2, hipTextureObject_t sinc_lut, float lut_center) {
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	int m = blockIdx.y * blockDim.y + threadIdx.y;
	int n_ini = blockIdx.z * ini_red;
	int n_max = fminf(n_ini + ini_red, N);
	
	if (m<M && t<T) {
		scalar_t loc_sum = 0;
		for (int n=n_ini; n<n_max; n++) {
			loc_sum += image_sample(amp[m*N+n], tau[m*N+n], t, Tw_2, sinc_lut, lut_center);
		}
		initialRIR[m*T*iniRIR_N + t*iniRIR_N + blockIdx.z] = loc_sum;
	}
}

__global__ void reduceRIR_kernel(scalar_t* initialRIR, scalar_t* intermediateRIR, int M, int T, int N, int intRIR_N) {
	extern __shared__ scalar_t sdata[];
	
	int tid = threadIdx.x;
	int n = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	int t = blockIdx.y;
	int m = blockIdx.z;
	
	if (n+blockDim.x < N) sdata[tid] = initialRIR[m*T*N + t*N + n] + initialRIR[m*T*N + t*N + n+blockDim.x];
	else if (n<N) sdata[tid] = initialRIR[m*T*N + t*N + n];
	else sdata[tid] = 0;
	__syncthreads();
	
	for (int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) sdata[tid] += sdata[tid+s];
		__syncthreads();
	}
	
	if (tid==0) {
		intermediateRIR[m*T*intRIR_N + t*intRIR_N + blockIdx.x] = sdata[0];
	}
}

__global__ void envPred_kernel(scalar_t* A /*[M_src]M_rcv]*/, scalar_t* alpha /*[M_src]M_rcv]*/, 
						scalar_t* RIRs_early /*[M_src][M_rcv][nSamples]*/, scalar_t* tau_dp, /*[M_src]M_rcv]*/
						int M_src, int M_rcv, int nSamples, scalar_t Fs,
						scalar_t room_sz_x, scalar_t room_sz_y, scalar_t room_sz_z,
						scalar_t beta_x1, scalar_t beta_x2, scalar_t beta_y1, scalar_t beta_y2, scalar_t beta_z1, scalar_t beta_z2) {
		
	scalar_t w_sz = 10e-3f * Fs; // Maximum window size (samples) to compute the final power of the early RIRs_early
	
	int m_src = blockIdx.x * blockDim.x + threadIdx.x;
	int m_rcv = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (m_src<M_src && m_rcv<M_rcv) {
		int w_start = __float2int_ru( max(nSamples-w_sz, tau_dp[m_src*M_rcv+m_rcv]));
		scalar_t w_center = (w_start + (nSamples-w_start)/2.0);
		
		scalar_t finalPower = 0.0f;
		for (int t=w_start; t<nSamples; t++) {
			scalar_t aux = RIRs_early[m_src*M_rcv*nSamples + m_rcv*nSamples + t];
			finalPower += aux*aux;
		}
		finalPower /= nSamples-w_start;
		
		scalar_t T60 = SabineT60(room_sz_x, room_sz_y, room_sz_z, beta_x1, beta_x2, beta_y1, beta_y2, beta_z1, beta_z2);
		scalar_t loc_alpha = -13.8155f / (T60 * Fs); //-13.8155 == log(10^(-6))
		
		A[m_src*M_rcv + m_rcv] = finalPower / expf(loc_alpha*(w_center-tau_dp[m_src*M_rcv+m_rcv]));
		alpha[m_src*M_rcv + m_rcv] = loc_alpha;
	}
}

__global__ void diffRev_kernel(scalar_t* rir, scalar_t* A, scalar_t* alpha, scalar_t* tau_dp,
							   int M_src, int M_rcv, int nSamplesISM, int nSamplesDiff) {
	
	int sample = blockIdx.x * blockDim.x + threadIdx.x;
	int m_src  = blockIdx.y * blockDim.y + threadIdx.y;
	int m_rcv  = blockIdx.z * blockDim.z + threadIdx.z;
	
	if (sample<nSamplesDiff && m_src<M_src && m_rcv<M_rcv) {
		// Get logistic distribution from uniform distribution
		scalar_t uniform = rir[m_src*M_rcv*nSamplesDiff + m_rcv*nSamplesDiff + sample];
		scalar_t logistic = 0.551329f * logf(uniform/(1.0f - uniform + 1e-6)); // 0.551329 == sqrt(3)/pi
		
		// Apply power envelope
		scalar_t pow_env = A[m_src*M_rcv+m_rcv] * expf(alpha[m_src*M_rcv+m_rcv] * (nSamplesISM+sample-tau_dp[m_src*M_rcv+m_rcv]));
		rir[m_src*M_rcv*nSamplesDiff + m_rcv*nSamplesDiff + sample] = sqrt(pow_env) * logistic;
	}
}

__global__ void complexPointwiseMulAndScale(hipfftComplex *signal_segments, hipfftComplex *RIRs, int segment_size, int M_rcv, int M_src, float scale) {
    int numThreads_x = blockDim.x * gridDim.x;
    int numThreads_y = blockDim.y * gridDim.y;
    int numThreads_z = blockDim.z * gridDim.z;
	
    int threadID_x = blockIdx.x * blockDim.x + threadIdx.x;
    int threadID_y = blockIdx.y * blockDim.y + threadIdx.y;
    int threadID_z = blockIdx.z * blockDim.z + threadIdx.z;

	for (int m = threadID_z; m < M_src; m += numThreads_z) {
		for (int n = threadID_y; n < M_rcv; n += numThreads_y) {
			for (int i = threadID_x; i < segment_size; i += numThreads_x) {
				RIRs[m*M_rcv*segment_size + n*segment_size + i] = 
					ComplexScale(ComplexMul(RIRs[m*M_rcv*segment_size + n*segment_size + i], 
											signal_segments[m*segment_size + i]), 
								 scale);
			}
		}
	}
}

/***************************/
/* Mixed precision KERNELS */
/***************************/

#if CUDART_VERSION < 9020
__global__ void generateRIR_mp_kernel(half2* initialRIR, scalar_t* amp, scalar_t* tau, int T, int M, int N, int iniRIR_N, int ini_red, scalar_t Fs, scalar_t Tw_2, scalar_t Tw_inv) {
	half2 h2Tw_inv = __float2half2_rn(Tw_inv);
#else 
__global__ void generateRIR_mp_kernel(half2* initialRIR, scalar_t* amp, scalar_t* tau, int T, int M, int N, int iniRIR_N, int ini_red, scalar_t Fs, scalar_t Tw_2, half2 h2Tw_inv) {
#endif
	#if __CUDA_ARCH__ >= 530
		int t = blockIdx.x * blockDim.x + threadIdx.x;
		int m = blockIdx.y * blockDim.y + threadIdx.y;
		int n_ini = blockIdx.z * ini_red;
		int n_max = fminf(n_ini + ini_red, N);
		
		if (m<M && t<T) {
			half2 loc_sum = h2zeros;
			scalar_t loc_tim_1 = 2*t;
			scalar_t loc_tim_2 = 2*t+1;
			for (int n=n_ini; n<n_max; n++) {
				half2 amp_mp = __float2half2_rn(amp[m*N+n]);
				loc_sum = __hadd2(loc_sum, image_sample_mp(amp_mp, tau[m*N+n], loc_tim_1, loc_tim_2, Tw_2, h2Tw_inv));
			}
			initialRIR[m*T*iniRIR_N + t*iniRIR_N + blockIdx.z] = loc_sum;
		}
	#else
		printf("Mixed precision requires Pascal GPU architecture or higher.\n");
	#endif
}

__global__ void reduceRIR_mp_kernel(half2* initialRIR, half2* intermediateRIR, int M, int T, int N, int intRIR_N) {
	extern __shared__ half2 sdata_mp[];
	#if __CUDA_ARCH__ >= 530
		int tid = threadIdx.x;
		int n = blockIdx.x*(blockDim.x*2) + threadIdx.x;
		int t = blockIdx.y;
		int m = blockIdx.z;

		if (n+blockDim.x < N) sdata_mp[tid] = __hadd2(initialRIR[m*T*N + t*N + n], initialRIR[m*T*N + t*N + n+blockDim.x]);
		else if (n<N) sdata_mp[tid] = initialRIR[m*T*N + t*N + n];
		else sdata_mp[tid] = h2zeros;
		__syncthreads();

		for (int s=blockDim.x/2; s>0; s>>=1) {
			if (tid < s) sdata_mp[tid] = __hadd2(sdata_mp[tid], sdata_mp[tid+s]);
			__syncthreads();
		}

		if (tid==0) {
			intermediateRIR[m*T*intRIR_N + t*intRIR_N + blockIdx.x] = sdata_mp[0];
		}
	#else
		printf("Mixed precision requires Pascal GPU architecture or higher.\n");
	#endif
}

__global__ void h2RIR_to_floatRIR_kernel(half2* h2RIR, scalar_t* floatRIR, int M, int T) {
	#if __CUDA_ARCH__ >= 530
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	int m = blockIdx.y * blockDim.y + threadIdx.y;

	if (t<T && m<M) {
		floatRIR[m*2*T + 2*t  ] =  __low2float(h2RIR[m*T + t]);
		floatRIR[m*2*T + 2*t+1] = __high2float(h2RIR[m*T + t]);
	}
	#else
		printf("Mixed precision requires Pascal GPU architecture or higher.\n");
	#endif
}

/***************************/
/* Auxiliar host functions */
/***************************/

hipTextureObject_t create_sinc_texture_lut(hipArray **cuArrayLut, int Tw, int lut_len) {
	// Create lut in host memory
	int lut_center = lut_len / 2;
	scalar_t* sinc_lut_host = (scalar_t*)malloc(sizeof(scalar_t) * lut_len);
	for (int i=0; i<=lut_center; i++) {
		scalar_t x = (float)i / lut_oversamp;
		scalar_t sinc = (x==0.0f)? 1.0f : sin(PI*x) / (PI*x);
		scalar_t hann = 0.5f * (1.0f + cos(2.0f*PI*x/Tw));
		scalar_t y = hann * sinc;
		sinc_lut_host[lut_center+i] = y;
		sinc_lut_host[lut_center-i] = y;
	}
	
	// Copy the lut to a device hipArray
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipMallocArray(cuArrayLut, &channelDesc, lut_len);
    hipMemcpyToArray(*cuArrayLut, 0, 0, sinc_lut_host, sizeof(scalar_t)*lut_len,
                      hipMemcpyHostToDevice);
	
	// Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = *cuArrayLut;

	// Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeBorder;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
	
	// Create texture object
    hipTextureObject_t texObjLut = 0;
    hipCreateTextureObject(&texObjLut, &resDesc, &texDesc, NULL);
	
	return texObjLut;
}

void gpuRIR_cuda::cuda_rirGenerator(scalar_t* rir, scalar_t* amp, scalar_t* tau, int M, int N, int T, scalar_t Fs) {
	int initialReduction = initialReductionMin;
	while (M * T * ceil((float)N/initialReduction) > 1e9) initialReduction *= 2;
	
	int iniRIR_N = ceil((float)N/initialReduction);
	dim3 threadsPerBlockIni(nThreadsGen_t, nThreadsGen_m, nThreadsGen_n);
	dim3 numBlocksIni(ceil((float)T/threadsPerBlockIni.x), ceil((float)M/threadsPerBlockIni.y), iniRIR_N);
	
	scalar_t* initialRIR;
	gpuErrchk( hipMalloc(&initialRIR, M*T*iniRIR_N*sizeof(scalar_t)) );
	
	int Tw = (int) round(8e-3f * Fs); // Window duration [samples]
	int lut_len = Tw * lut_oversamp;
	lut_len += ((lut_len%2)? 0 : 1); // Must be odd
	hipArray* cuArrayLut;
	hipTextureObject_t sinc_lut = create_sinc_texture_lut(&cuArrayLut, Tw, lut_len);
	
	generateRIR_kernel<<<numBlocksIni, threadsPerBlockIni>>>( initialRIR, amp, tau, T, M, N, iniRIR_N, initialReduction, Tw/2, sinc_lut, lut_len/2+0.5 );
	gpuErrchk( hipDeviceSynchronize() );
	gpuErrchk( hipPeekAtLastError() );
	hipDestroyTextureObject(sinc_lut);
	hipFreeArray(cuArrayLut);
	
	dim3 threadsPerBlockRed(nThreadsRed, 1, 1);
	scalar_t* intermediateRIR;
	int intRIR_N;
	while (iniRIR_N > 2*nThreadsRed) {		
		intRIR_N = ceil((float)iniRIR_N / (2*nThreadsRed));
		gpuErrchk( hipMalloc(&intermediateRIR, intRIR_N * T * M * sizeof(scalar_t)) );

		dim3 numBlocksRed(intRIR_N, T, M);
		reduceRIR_kernel<<<numBlocksRed, threadsPerBlockRed, nThreadsRed*sizeof(scalar_t)>>>(
			initialRIR, intermediateRIR, M, T, iniRIR_N, intRIR_N);
		gpuErrchk( hipDeviceSynchronize() );
		gpuErrchk( hipPeekAtLastError() );
		
		gpuErrchk( hipFree(initialRIR) );
		initialRIR = intermediateRIR;		
		iniRIR_N = intRIR_N;
	}
	
	dim3 numBlocksEnd(1, T, M);
	reduceRIR_kernel<<<numBlocksEnd, threadsPerBlockRed, nThreadsRed*sizeof(scalar_t)>>>(
		initialRIR, rir, M, T, iniRIR_N, 1);
	gpuErrchk( hipDeviceSynchronize() );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(initialRIR) );
}

void cuda_rirGenerator_mp(scalar_t* rir, scalar_t* amp, scalar_t* tau, int M, int N, int T, scalar_t Fs) {
	if (cuda_arch >= 530) {
		int initialReduction = initialReductionMin;
		while (M * T/2 * ceil((float)N/initialReduction) > 1e9) initialReduction *= 2;

		int iniRIR_N = ceil((float)N/initialReduction);
		dim3 threadsPerBlockIni(nThreadsGen_t, nThreadsGen_m, nThreadsGen_n);
		dim3 numBlocksIni(ceil((float)T/2/threadsPerBlockIni.x), ceil((float)M/threadsPerBlockIni.y), iniRIR_N);

		half2* initialRIR;
		gpuErrchk( hipMalloc(&initialRIR, M*(T/2)*iniRIR_N*sizeof(half2)) );

		scalar_t Tw_2 = 8e-3f * Fs / 2;
		#if CUDART_VERSION < 9020
			// For CUDA versions older than 9.2 it is nos possible to call from host code __float2half2_rn,
			// but doing it in the kernel is slower
			scalar_t Tw_inv = 1.0f / (8e-3f * Fs);
		#else 
			half2 Tw_inv = __float2half2_rn(1.0f / (8e-3f * Fs));
		#endif
		generateRIR_mp_kernel<<<numBlocksIni, threadsPerBlockIni>>>( initialRIR, amp, tau, T/2, M, N, iniRIR_N, initialReduction, Fs, Tw_2, Tw_inv );
		gpuErrchk( hipDeviceSynchronize() );
		gpuErrchk( hipPeekAtLastError() );

		dim3 threadsPerBlockRed(nThreadsRed, 1, 1);
		half2* intermediateRIR;
		int intRIR_N;
		while (iniRIR_N > 2*nThreadsRed) {
			intRIR_N = ceil((float)iniRIR_N / (2*nThreadsRed));
			gpuErrchk( hipMalloc(&intermediateRIR, intRIR_N * T/2 * M * sizeof(half2)) );

			dim3 numBlocksRed(intRIR_N, T/2, M);
			reduceRIR_mp_kernel<<<numBlocksRed, threadsPerBlockRed, nThreadsRed*sizeof(half2)>>>(
				initialRIR, intermediateRIR, M, T/2, iniRIR_N, intRIR_N);
			gpuErrchk( hipDeviceSynchronize() );
			gpuErrchk( hipPeekAtLastError() );

			gpuErrchk( hipFree(initialRIR) );
			initialRIR = intermediateRIR;
			iniRIR_N = intRIR_N;
		}

		gpuErrchk( hipMalloc(&intermediateRIR, M * T/2 * sizeof(half2)) );
		dim3 numBlocksEnd(1, T/2, M);
		reduceRIR_mp_kernel<<<numBlocksEnd, threadsPerBlockRed, nThreadsRed*sizeof(half2)>>>(
			initialRIR, intermediateRIR, M, T/2, iniRIR_N, 1);
		gpuErrchk( hipDeviceSynchronize() );
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipFree(initialRIR) );

		dim3 numBlocks(ceil((float)(T/2)/128), M, 1);
		dim3 threadsPerBlock(128, 1, 1);
		h2RIR_to_floatRIR_kernel<<<numBlocks, threadsPerBlock>>>(intermediateRIR, rir, M, T/2);

		gpuErrchk( hipFree(intermediateRIR) );
	} else {
		printf("Mixed precision requires Pascal GPU architecture or higher.\n");
	}
}

int gpuRIR_cuda::PadData(scalar_t *signal, scalar_t **padded_signal, int segment_len,
						 scalar_t *RIR, scalar_t **padded_RIR, int M_src, int M_rcv, int RIR_len) {
				
    int N_fft = pow2roundup(segment_len + RIR_len - 1);

    // Pad signal
    float *new_data = (float *)malloc(sizeof(float) * M_src * (N_fft+2));
	for (int m=0; m<M_src; m++) {
		memcpy(new_data + m*(N_fft+2), signal + m*segment_len, segment_len*sizeof(float));
		memset(new_data + m*(N_fft+2) + segment_len, 0, ((N_fft+2)-segment_len)*sizeof(float));
	}
    *padded_signal = new_data;

    // Pad filter
    new_data = (float *)malloc(sizeof(float) * M_src * M_rcv * (N_fft+2));
	for (int m=0; m<M_src; m++) {
		for (int n=0; n<M_rcv; n++) {
			memcpy(new_data + m*M_rcv*(N_fft+2) + n*(N_fft+2), RIR + m*M_rcv*RIR_len + n*RIR_len, RIR_len*sizeof(float));
			memset(new_data + m*M_rcv*(N_fft+2) + n*(N_fft+2) + RIR_len, 0, ((N_fft+2)-RIR_len)*sizeof(float));
		}
	}
    *padded_RIR = new_data;

    return N_fft;
}

/***********************/
/* Principal functions */
/***********************/

scalar_t* gpuRIR_cuda::cuda_simulateRIR(scalar_t room_sz[3], scalar_t beta[6], scalar_t* h_pos_src, int M_src, 
									   scalar_t* h_pos_rcv, scalar_t* h_orV_rcv, micPattern mic_pattern, int M_rcv, int nb_img[3],
									   scalar_t Tdiff, scalar_t Tmax, scalar_t Fs, scalar_t c) {	
	// function scalar_t* cuda_simulateRIR(scalar_t room_sz[3], scalar_t beta[6], scalar_t* h_pos_src, int M_src, 
	//									   scalar_t* h_pos_rcv, scalar_t* h_orV_rcv, micPattern mic_pattern, int M_rcv, int nb_img[3],
	//									   scalar_t Tdiff, scalar_t Tmax, scalar_t Fs, scalar_t c);
	// Input parameters:
	// 	scalar_t room_sz[3]		: Size of the room [m]
	//	scalar_t beta[6] 		: Reflection coefficients [beta_x1 beta_x2 beta_y1 beta_y2 beta_z1 beta_z2]
	//	scalar_t* h_pos_src 	: M_src x 3 matrix with the positions of the sources [m]
	//	int M_src 				: Number of sources
	//	scalar_t* h_pos_rcv 	: M_rcv x 3 matrix with the positions of the receivers [m]
	//	scalar_t* h_orV_rcv 	: M_rcv x 3 matrix with vectors pointing in the same direction than the receivers
	//	micPattern mic_pattern 	: Polar pattern of the receivers (see gpuRIR_cuda.h)
	//	int M_rcv 				: Number of receivers
	//	int nb_img[3] 			: Number of sources in each dimension
	//	scalar_t Tdiff			: Time when the ISM is replaced by a diffusse reverberation model [s]
	//	scalar_t Tmax 			: RIRs length [s]
	//	scalar_t Fs				: Sampling frequency [Hz]
	//	scalar_t c				: Speed of sound [m/s]
	
	// Copy host memory to GPU
	scalar_t *pos_src, *pos_rcv, *orV_rcv;
	gpuErrchk( hipMalloc(&pos_src, M_src*3*sizeof(scalar_t)) );
	gpuErrchk( hipMalloc(&pos_rcv, M_rcv*3*sizeof(scalar_t)) );
	gpuErrchk( hipMalloc(&orV_rcv, M_rcv*3*sizeof(scalar_t)) );
	gpuErrchk( hipMemcpy(pos_src, h_pos_src, M_src*3*sizeof(scalar_t), hipMemcpyHostToDevice ) );
	gpuErrchk( hipMemcpy(pos_rcv, h_pos_rcv, M_rcv*3*sizeof(scalar_t), hipMemcpyHostToDevice ) );
	gpuErrchk( hipMemcpy(orV_rcv, h_orV_rcv, M_rcv*3*sizeof(scalar_t), hipMemcpyHostToDevice ) );
	
	
	// Use the ISM to calculate the amplitude and delay of each image
	dim3 threadsPerBlockISM(nThreadsISM_x, nThreadsISM_y, nThreadsISM_z);
	dim3 numBlocksISM(ceil((float)nb_img[0] / nThreadsISM_x), 
					  ceil((float)nb_img[1] / nThreadsISM_y), 
					  ceil((float)nb_img[2] / nThreadsISM_z));
	int shMemISM = (M_src + 2*M_rcv) * 3 * sizeof(scalar_t);
	
	scalar_t* amp; // Amplitude with which the signals from each image source of each source arrive to each receiver
	gpuErrchk( hipMalloc(&amp, M_src*M_rcv*nb_img[0]*nb_img[1]*nb_img[2]*sizeof(scalar_t)) );
	scalar_t* tau; // Delay with which the signals from each image source of each source arrive to each receiver
	gpuErrchk( hipMalloc(&tau, M_src*M_rcv*nb_img[0]*nb_img[1]*nb_img[2]*sizeof(scalar_t)) );
	scalar_t* tau_dp; // Direct path delay
	gpuErrchk( hipMalloc(&tau_dp, M_src*M_rcv*sizeof(scalar_t)) );
	
	calcAmpTau_kernel<<<numBlocksISM, threadsPerBlockISM, shMemISM>>> (
		amp, tau, tau_dp,
		pos_src, pos_rcv, orV_rcv, mic_pattern,
		room_sz[0], room_sz[1], room_sz[2], 
		beta[0], beta[1], beta[2], beta[3], beta[4], beta[5], 
		nb_img[0], nb_img[1], nb_img[2],
		M_src, M_rcv, c, Fs
	);
	gpuErrchk( hipDeviceSynchronize() );
	gpuErrchk( hipPeekAtLastError() );

	int nSamplesISM = ceil(Tdiff*Fs);
	nSamplesISM += nSamplesISM%2; // nSamplesISM must be even
	int nSamples = ceil(Tmax*Fs);
	nSamples += nSamples%2; // nSamples must be even
	int nSamplesDiff = nSamples - nSamplesISM;
	
	// Compute the RIRs as a sum of sincs
	int M = M_src * M_rcv;
	int N = nb_img[0] * nb_img[1] * nb_img[2];
	scalar_t* rirISM;
	gpuErrchk( hipMalloc(&rirISM, M*nSamplesISM*sizeof(scalar_t)) );
	if (mixed_precision) {
		if (cuda_arch >= 530) {
			cuda_rirGenerator_mp(rirISM, amp, tau, M, N, nSamplesISM, Fs);
		} else {
			printf("The mixed precision requires Pascal GPU architecture or higher.\n");
		}
	} else {
		cuda_rirGenerator(rirISM, amp, tau, M, N, nSamplesISM, Fs);
	}
	
	// Compute the exponential power envelope parammeters of each RIR
	dim3 threadsPerBlockEnvPred(nThreadsEnvPred_x, nThreadsEnvPred_y, nThreadsEnvPred_z);
	dim3 numBlocksEnvPred(ceil((float)M_src / nThreadsEnvPred_x), 
						  ceil((float)M_rcv / nThreadsEnvPred_y), 1);
					  
	scalar_t* A; // pow_env = A * exp(alpha * (t-tau_dp))
	gpuErrchk( hipMalloc(&A, M_src*M_rcv*sizeof(scalar_t)) );
	scalar_t* alpha;
	gpuErrchk( hipMalloc(&alpha, M_src*M_rcv*sizeof(scalar_t)) );
	
	envPred_kernel<<<numBlocksEnvPred, threadsPerBlockEnvPred>>>(
			A, alpha, rirISM, tau_dp, M_src, M_rcv, nSamplesISM, Fs,
			room_sz[0], room_sz[1], room_sz[2], beta[0], beta[1], beta[2], beta[3], beta[4], beta[5]);
	gpuErrchk( hipDeviceSynchronize() );
	gpuErrchk( hipPeekAtLastError() );
	
	// Generate diffuse reverberation
	scalar_t* rirDiff; 
	gpuErrchk( hipMalloc(&rirDiff, M_src*M_rcv*nSamplesDiff*sizeof(scalar_t)) );
	
	if (nSamplesDiff != 0) {
		// Fill rirDiff with random numbers with uniform distribution
		gpuErrchk( hiprandGenerateUniform(cuRandGenWrap.gen, rirDiff, M_src*M_rcv*nSamplesDiff) );
		gpuErrchk( hipDeviceSynchronize() );
		gpuErrchk( hipPeekAtLastError() );
				
		dim3 threadsPerBlockDiff(nThreadsDiff_t, nThreadsDiff_src, nThreadsDiff_rcv);
		dim3 numBlocksDiff(ceil((float)nSamplesDiff / nThreadsDiff_t),
							  ceil((float)M_src / nThreadsDiff_src), 
							  ceil((float)M_rcv / nThreadsDiff_rcv));
		diffRev_kernel<<<numBlocksDiff, threadsPerBlockDiff>>>(
				rirDiff, A, alpha, tau_dp, M_src, M_rcv, nSamplesISM, nSamplesDiff);
		gpuErrchk( hipDeviceSynchronize() );
		gpuErrchk( hipPeekAtLastError() );
	}
	
	// Copy GPU memory to host
	int rirSizeISM = M_src * M_rcv * nSamplesISM * sizeof(scalar_t);
	int rirSizeDiff = M_src * M_rcv * nSamplesDiff * sizeof(scalar_t);
	scalar_t* h_rir = (scalar_t*) malloc(rirSizeISM+rirSizeDiff);
	
	hipPitchedPtr h_rir_pitchedPtr = make_hipPitchedPtr( (void*) h_rir, 
		(nSamplesISM+nSamplesDiff)*sizeof(scalar_t), nSamplesISM+nSamplesDiff, M_rcv );
	hipPitchedPtr rirISM_pitchedPtr = make_hipPitchedPtr( (void*) rirISM, 
		nSamplesISM*sizeof(scalar_t), nSamplesISM, M_rcv );
	hipPitchedPtr rirDiff_pitchedPtr = make_hipPitchedPtr( (void*) rirDiff, 
		nSamplesDiff*sizeof(scalar_t), nSamplesDiff, M_rcv );
	
	hipMemcpy3DParms parmsISM = {0};
	parmsISM.srcPtr = rirISM_pitchedPtr;
	parmsISM.dstPtr = h_rir_pitchedPtr;
	parmsISM.extent = make_hipExtent(nSamplesISM*sizeof(scalar_t), M_rcv, M_src);
	parmsISM.kind = hipMemcpyDeviceToHost;
	gpuErrchk( hipMemcpy3D(&parmsISM) );
	
	if (nSamplesDiff > 0) {
		hipMemcpy3DParms parmsDiff = {0};
		parmsDiff.srcPtr = rirDiff_pitchedPtr;
		parmsDiff.dstPtr = h_rir_pitchedPtr;
		parmsDiff.dstPos = make_hipPos(nSamplesISM*sizeof(scalar_t), 0, 0);
		parmsDiff.extent = make_hipExtent(nSamplesDiff*sizeof(scalar_t), M_rcv, M_src);
		parmsDiff.kind = hipMemcpyDeviceToHost;
		gpuErrchk( hipMemcpy3D(&parmsDiff) );
	}

	// Free memory
	gpuErrchk( hipFree(pos_src) );
	gpuErrchk( hipFree(pos_rcv) );
	gpuErrchk( hipFree(orV_rcv) );
	gpuErrchk( hipFree(amp)	 );
	gpuErrchk( hipFree(tau)	 );
	gpuErrchk( hipFree(tau_dp)	 );
	gpuErrchk( hipFree(rirISM)	 );
	gpuErrchk( hipFree(A)		 );
	gpuErrchk( hipFree(alpha)	 );
	gpuErrchk( hipFree(rirDiff) );
	
	return h_rir;
}

scalar_t* gpuRIR_cuda::cuda_convolutions(scalar_t* source_segments, int M_src, int segment_len,
										scalar_t* RIR, int M_rcv, int RIR_len) {	
	// function scalar_t* cuda_filterRIR(scalar_t* source_segments, int M_src, int segments_len,
	//									 scalar_t* RIR, int M_rcv, int RIR_len);
	// Input parameters:
	// 	scalar_t* source_segments : Source signal segment for each trajectory point
	//	int M_src 				  : Number of trajectory points
	//	int segment_len 		  : Length of the segments [samples]
	//	scalar_t* RIR		 	  : 3D array with the RIR from each point of the trajectory to each receiver
	//	int M_rcv 				  : Number of receivers
	//	int RIR_len 			  : Length of the RIRs [samples]

	// Size of the FFT needed to avoid circular convolution effects
	int N_fft = pow2roundup(segment_len + RIR_len - 1);
	
	// Copy the signal segments with zero padding
    int mem_size_signal = sizeof(scalar_t) * M_src * (N_fft+2);
    hipfftComplex *d_signal;
    gpuErrchk( hipMalloc((void **)&d_signal, mem_size_signal) );
	gpuErrchk( hipMemcpy2D((void *)d_signal, (N_fft+2)*sizeof(scalar_t), 
		(void *)source_segments, segment_len*sizeof(scalar_t),
		segment_len*sizeof(scalar_t), M_src, hipMemcpyHostToDevice) );
	gpuErrchk( hipMemset2D((void *)((scalar_t *)d_signal + segment_len), (N_fft+2)*sizeof(scalar_t),
		0, (N_fft+2-segment_len)*sizeof(scalar_t), M_src ) );
	
	// Copy the RIRs with zero padding
	hipPitchedPtr h_RIR_pitchedPtr = make_hipPitchedPtr( (void*) RIR, 
		RIR_len*sizeof(scalar_t), RIR_len, M_rcv );
    int mem_size_RIR = sizeof(scalar_t) * M_src * M_rcv * (N_fft+2);
	hipfftComplex *d_RIR;
	gpuErrchk( hipMalloc((void **)&d_RIR, mem_size_RIR) );
	hipPitchedPtr d_RIR_pitchedPtr = make_hipPitchedPtr( (void*) d_RIR, 
		(N_fft+2)*sizeof(scalar_t), (N_fft+2), M_rcv );
	hipMemcpy3DParms parmsCopySignal = {0};
	parmsCopySignal.srcPtr = h_RIR_pitchedPtr;
	parmsCopySignal.dstPtr = d_RIR_pitchedPtr;
	parmsCopySignal.extent = make_hipExtent(RIR_len*sizeof(scalar_t), M_rcv, M_src);
	parmsCopySignal.kind = hipMemcpyHostToDevice;
	gpuErrchk( hipMemcpy3D(&parmsCopySignal) );
	gpuErrchk( hipMemset2D((void *)((scalar_t *)d_RIR + RIR_len), (N_fft+2)*sizeof(scalar_t),
		0, (N_fft+2-RIR_len)*sizeof(scalar_t), M_rcv*M_src ) );
	
	// CUFFT plans
    hipfftHandle plan_signal, plan_RIR, plan_RIR_inv;
    gpuErrchk( hipfftPlan1d(&plan_signal,  N_fft, HIPFFT_R2C, M_src) );
    gpuErrchk( hipfftPlan1d(&plan_RIR,     N_fft, HIPFFT_R2C, M_src * M_rcv) );
    gpuErrchk( hipfftPlan1d(&plan_RIR_inv, N_fft, HIPFFT_C2R, M_src * M_rcv) );
	
	// Transform signal and RIR
    gpuErrchk( hipfftExecR2C(plan_signal, (hipfftReal *)d_signal, (hipfftComplex *)d_signal) );
    gpuErrchk( hipfftExecR2C(plan_RIR,    (hipfftReal *)d_RIR,    (hipfftComplex *)d_RIR   ) );
	
	// Multiply the coefficients together and normalize the result
	dim3 threadsPerBlock(nThreadsConv_x, nThreadsConv_y, nThreadsConv_z);
	int numBlocks_x = (int) ceil((float)(N_fft/2+1)/nThreadsConv_x);
	int numBlocks_y = (int) ceil((float)M_rcv/nThreadsConv_y);
	int numBlocks_z = (int) ceil((float)M_src/nThreadsConv_z);
	dim3 numBlocks(numBlocks_x, numBlocks_y, numBlocks_z);
    complexPointwiseMulAndScale<<<numBlocks, threadsPerBlock>>>
			(d_signal, d_RIR, (N_fft/2+1), M_rcv, M_src, 1.0f/N_fft);
	gpuErrchk( hipDeviceSynchronize() );
	gpuErrchk( hipPeekAtLastError() );
	
	// Transform signal back
    gpuErrchk( hipfftExecC2R(plan_RIR_inv, (hipfftComplex *)d_RIR, (hipfftReal *)d_RIR) );
	
	// Copy device memory to host
	int conv_len = segment_len + RIR_len - 1;
    scalar_t *convolved_segments = (scalar_t *)malloc(sizeof(scalar_t)*M_src*M_rcv*conv_len);
	hipPitchedPtr d_convolved_segments_pitchedPtr = make_hipPitchedPtr( (void*) d_RIR, 
		(N_fft+2)*sizeof(scalar_t), conv_len, M_rcv );
	hipPitchedPtr h_convolved_segments_pitchedPtr = make_hipPitchedPtr( (void*) convolved_segments, 
		conv_len*sizeof(scalar_t), conv_len, M_rcv );
	hipMemcpy3DParms parmsCopy = {0};
	parmsCopy.srcPtr = d_convolved_segments_pitchedPtr;
	parmsCopy.dstPtr = h_convolved_segments_pitchedPtr;
	parmsCopy.extent = make_hipExtent(conv_len*sizeof(scalar_t), M_rcv, M_src);
	parmsCopy.kind = hipMemcpyDeviceToHost;
	gpuErrchk( hipMemcpy3D(&parmsCopy) );

	//Destroy CUFFT context
    gpuErrchk( hipfftDestroy(plan_signal) );
    gpuErrchk( hipfftDestroy(plan_RIR) );
    gpuErrchk( hipfftDestroy(plan_RIR_inv) );

    // cleanup memory
    gpuErrchk( hipFree(d_signal) );
    gpuErrchk( hipFree(d_RIR) );
	
	return convolved_segments;
}

gpuRIR_cuda::gpuRIR_cuda(bool mPrecision) {
	// Get CUDA architecture
	hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
	cuda_arch = prop.major*100 + prop.minor*10;

	// Activate mixed precision if selected
	activate_mixed_precision(mPrecision);
	
	// Initiate CUDA runtime API
	scalar_t* memPtr_warmup;
	gpuErrchk( hipMalloc(&memPtr_warmup, 1*sizeof(scalar_t)) );
	gpuErrchk( hipFree(memPtr_warmup) );
	
	// Initiate cuFFT library
	hipfftHandle plan_warmup;
	gpuErrchk( hipfftPlan1d(&plan_warmup,  1024, HIPFFT_R2C, 1) );
	gpuErrchk( hipfftDestroy(plan_warmup) );

	// Initialize cuRAND generator
	gpuErrchk( hiprandCreateGenerator(&cuRandGenWrap.gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
	gpuErrchk( hiprandSetPseudoRandomGeneratorSeed(cuRandGenWrap.gen, 1234ULL) );
}

bool gpuRIR_cuda::activate_mixed_precision(bool activate) {
	if (cuda_arch >= 530) {
		mixed_precision = activate;
	} else {
		if (activate) printf("This feature requires Pascal GPU architecture or higher.\n");
		mixed_precision = false;
	}
	return mixed_precision;
}
