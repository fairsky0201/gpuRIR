
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#include "/usr/local/cuda/include/cuda.h"
#include "/usr/local/cuda/include/cuda_runtime.h"
#include <hiprand/hiprand.h>

#include <vector>

#define PI 3.141592654f

typedef float scalar_t;

/******************************/
/* Parallelization parameters */
/******************************/

// Image Source Method
const int nThreadsISM_x = 4;
const int nThreadsISM_y = 4;
const int nThreadsISM_z = 4;

// Time vector generation
const int nThreadsTime = 128;

// RIR computation
const int initialReductionMin = 512;
const int nThreadsGen_t = 32;
const int nThreadsGen_m = 4;
const int nThreadsGen_n = 1; // Don't change it
const int nThreadsRed = 128;

// Power envelope prediction
const int nThreadsEnvPred_x = 4;
const int nThreadsEnvPred_y = 4;
const int nThreadsEnvPred_z = 1; // Don't change it

// Generate diffuse reverberation
const int nThreadsDiff_t = 16;
const int nThreadsDiff_src = 4;
const int nThreadsDiff_rcv = 2;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hiprandStatus_t code, const char *file, int line, bool abort=true) {
   if (code != HIPRAND_STATUS_SUCCESS) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", code, file, line);
      if (abort) exit(code);
   }
}


/*****************************/
/* Auxiliar device functions */
/*****************************/

__device__ __forceinline__ scalar_t hanning_window(scalar_t t, scalar_t Tw) {
	return 0.5f * (1.0f + __cosf(2.0f*PI*t/Tw));
}

__device__ __forceinline__ scalar_t sinc(scalar_t x) {
	return (x==0)? 1 : sinf(x)/x; 
}

__device__ __forceinline__ scalar_t image_sample(scalar_t amp, scalar_t tau, scalar_t t, scalar_t Fs) {
	scalar_t Tw = 8e-3f; // Window duration [s]
	return (abs(t-tau)<Tw/2)? hanning_window(t-tau, Tw) * amp * sinc( (t - tau) * Fs * PI ) : 0.0f;
}

__device__ __forceinline__ scalar_t SabineT60( scalar_t room_sz_x, scalar_t room_sz_y, scalar_t room_sz_z,
							scalar_t beta_x1, scalar_t beta_x2, scalar_t beta_y1, scalar_t beta_y2, scalar_t beta_z1, scalar_t beta_z2 ) {
	scalar_t Sa = ((1.0f-beta_x1*beta_x1) + (1.0f-beta_x2*beta_x2)) * room_sz_y * room_sz_z +
				  ((1.0f-beta_y1*beta_y1) + (1.0f-beta_y2*beta_y2)) * room_sz_x * room_sz_z +
				  ((1.0f-beta_z1*beta_z1) + (1.0f-beta_z2*beta_z2)) * room_sz_x * room_sz_y;
	scalar_t V = room_sz_x * room_sz_y * room_sz_z;
	return 0.161f * V / Sa;
}

/***********/
/* KERNELS */
/***********/

__global__ void calcAmpTau_kernel(scalar_t* g_amp /*[M_src]M_rcv][nb_img_x][nb_img_y][nb_img_z]*/, 
								  scalar_t* g_tau /*[M_src]M_rcv][nb_img_x][nb_img_y][nb_img_z]*/, 
								  scalar_t* g_tau_dp /*[M_src]M_rcv]*/,
								  scalar_t* g_pos_src/*[M_src][3]*/, scalar_t* g_pos_rcv/*[M_rcv][3]*/, 
								  scalar_t room_sz_x, scalar_t room_sz_y, scalar_t room_sz_z,
								  scalar_t beta_x1, scalar_t beta_x2, scalar_t beta_y1, scalar_t beta_y2, scalar_t beta_z1, scalar_t beta_z2, 
								  int nb_img_x, int nb_img_y, int nb_img_z,
								  int M_src, int M_rcv, scalar_t c) {	
	
	extern __shared__ scalar_t sdata[];
		
	int n[3];
	n[0] = blockIdx.x * blockDim.x + threadIdx.x;
	n[1] = blockIdx.y * blockDim.y + threadIdx.y;
	n[2] = blockIdx.z * blockDim.z + threadIdx.z;
	
	int N[3];
	N[0] = nb_img_x;
	N[1] = nb_img_y;
	N[2] = nb_img_z;
	
	scalar_t room_sz[3];
	room_sz[0] = room_sz_x;
	room_sz[1] = room_sz_y;
	room_sz[2] = room_sz_z;
	
	scalar_t beta[6];
	beta[0] = - beta_x1;
	beta[1] = - beta_x2;
	beta[2] = - beta_y1;
	beta[3] = - beta_y2;
	beta[4] = - beta_z1;
	beta[5] = - beta_z2;
	
	int prodN = N[0]*N[1]*N[2];
	int n_idx = n[0]*N[1]*N[2] + n[1]*N[2] + n[2];
	
	// Copy g_pos_src to shared memory
	scalar_t* sh_pos_src = (scalar_t*) sdata;
	if (threadIdx.y==0 && threadIdx.z==0)  {
		for (int m=threadIdx.x; m<M_src; m+=blockDim.x) {
			sh_pos_src[m*3  ] = g_pos_src[m*3  ];
			sh_pos_src[m*3+1] = g_pos_src[m*3+1];
			sh_pos_src[m*3+2] = g_pos_src[m*3+2];
		}
	}
	
	// Copy g_pos_rcv to shared memory
	scalar_t* sh_pos_rcv = &sh_pos_src[M_src*3];
	if (threadIdx.x==0 && threadIdx.z==0)  {
		for (int m=threadIdx.y; m<M_rcv; m+=blockDim.y) {
			sh_pos_rcv[m*3  ] = g_pos_rcv[m*3  ];
			sh_pos_rcv[m*3+1] = g_pos_rcv[m*3+1];
			sh_pos_rcv[m*3+2] = g_pos_rcv[m*3+2];
		}
	}
	
	// Wait until the copies are completed
	__syncthreads();
	
	if (n[0]<N[0] & n[1]<N[1] & n[2]<N[2]) {
		
		// Common factors for each src and rcv
		scalar_t rflx_att = 1;
		scalar_t clust_pos[3];
		int clust_idx[3];
		int rflx_idx[3];
		bool direct_path = true;
		for (int d=0; d<3; d++) {
			clust_idx[d] = __float2int_ru((n[d] - N[d]/2) / 2.0f); 
			clust_pos[d] = clust_idx[d] * 2*room_sz[d];
			rflx_idx[d] = abs((n[d] - N[d]/2) % 2); // 1 means reflected in dimension d
			rflx_att *= powf(beta[d*2], abs(clust_idx[d]-rflx_idx[d])) * powf(beta[d*2+1], abs(clust_idx[d]));
			direct_path *= (clust_idx[d]==0)&&(rflx_idx[d]==0);
		}
			
		// Individual factors for each src and rcv
		for (int m_src=0; m_src<M_src; m_src++) {
			for (int m_rcv=0; m_rcv<M_rcv; m_rcv++) {
				scalar_t vec[3];
				scalar_t dist = 0;
				for (int d=0; d<3; d++) {
					vec[d] = clust_pos[d] + (1-2*rflx_idx[d]) * sh_pos_src[m_src*3+d] - sh_pos_rcv[m_rcv*3+d];
					dist += vec[d] * vec[d];
				}
				dist = sqrtf(dist);
				//printf("%d, \n", m_src*M_rcv*prodN + m_rcv*prodN + n_idx);
				g_amp[m_src*M_rcv*prodN + m_rcv*prodN + n_idx] = rflx_att / (4*PI*dist);
				g_tau[m_src*M_rcv*prodN + m_rcv*prodN + n_idx] = dist / c;

				if (direct_path) g_tau_dp[m_src*M_rcv + m_rcv] = dist / c;
				
				//if (n[0] == 3) printf("%d %d %d %f %f\n", n[0], n[1], n[2], dist / c, rflx_att / (4*PI*dist));
			}
		}
	}
}

__global__ void generateTime_kernel(scalar_t* t, scalar_t Fs, int nSamples) {
	int sample = blockIdx.x * blockDim.x + threadIdx.x;
	if (sample<nSamples) {t[sample] = sample/Fs; /* printf("%d %f \n", sample, t[sample]); */} 
}

__global__ void generateRIR_kernel(scalar_t* initialRIR, scalar_t* tim, scalar_t* amp, scalar_t* tau, int T, int M, int N, int iniRIR_N, int ini_red, scalar_t Fs) {	
	int t = blockIdx.x * blockDim.x + threadIdx.x;
	int m = blockIdx.y * blockDim.y + threadIdx.y;
	int n_ini = blockIdx.z * ini_red;
	int n_max = fminf(n_ini + ini_red, N);
	
	if (m<M && t<T) {
		scalar_t loc_sum = 0;
		scalar_t loc_tim = tim[t];		
		for (int n=n_ini; n<n_max; n++) {
			loc_sum += image_sample(amp[m*N+n], tau[m*N+n], loc_tim, Fs);
			//if (t==19685) printf("%d %f %f %f %f\n", n, loc_tim, tau[m*N+n], amp[m*N+n], image_sample(amp[m*N+n], tau[m*N+n], loc_tim, Fs));
		}
		initialRIR[m*T*iniRIR_N + t*iniRIR_N + blockIdx.z] = loc_sum;
	}
}

__global__ void reduceRIR_kernel(scalar_t* initialRIR, scalar_t* intermediateRIR, int M, int T, int N, int intRIR_N) {
	extern __shared__ scalar_t sdata[];
	
	int tid = threadIdx.x;
	int n = blockIdx.x*(blockDim.x*2) + threadIdx.x;
	int t = blockIdx.y; //*blockDim.y + threadIdx.y;
	int m = blockIdx.z; //*blockDim.z + threadIdx.z;
	
	if (n+blockDim.x < N) sdata[tid] = initialRIR[m*T*N + t*N + n] + initialRIR[m*T*N + t*N + n+blockDim.x];
	else if (n<N) sdata[tid] = initialRIR[m*T*N + t*N + n];
	else sdata[tid] = 0;
	__syncthreads();
	
	for (int s=blockDim.x/2; s>0; s>>=1) {
		if (tid < s) sdata[tid] += sdata[tid+s];
		__syncthreads();
	}
	
	if (tid==0) {
		intermediateRIR[m*T*intRIR_N + t*intRIR_N + blockIdx.x] = sdata[0];
	}
}

__global__ void envPred_kernel(scalar_t* A /*[M_src]M_rcv]*/, scalar_t* alpha /*[M_src]M_rcv]*/, 
						scalar_t* RIRs_early /*[M_src][M_rcv][nSamples]*/, scalar_t* tau_dp, /*[M_src]M_rcv]*/
						int M_src, int M_rcv, int nSamples, scalar_t fs,
						scalar_t room_sz_x, scalar_t room_sz_y, scalar_t room_sz_z,
						scalar_t beta_x1, scalar_t beta_x2, scalar_t beta_y1, scalar_t beta_y2, scalar_t beta_z1, scalar_t beta_z2) {
		
	scalar_t w_sz = 10e-3f; // Maximum window size (s) to compute the final power of the early RIRs_early
	
	int m_src = blockIdx.x * blockDim.x + threadIdx.x;
	int m_rcv = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (m_src<M_src && m_rcv<M_rcv) {
		int w_start = __float2int_ru( max(nSamples/fs-w_sz, tau_dp[m_src*M_rcv+m_rcv]) * fs );
		scalar_t w_center = (w_start + (nSamples-w_start)/2.0) / fs;
		
		scalar_t finalPower = 0.0f;
		for (int t=w_start; t<nSamples; t++) {
			scalar_t aux = RIRs_early[m_src*M_rcv*nSamples + m_rcv*nSamples + t];
			finalPower += aux*aux;
		}
		finalPower /= nSamples-w_start;
		
		scalar_t T60 = SabineT60(room_sz_x, room_sz_y, room_sz_z, beta_x1, beta_x2, beta_y1, beta_y2, beta_z1, beta_z2);
		scalar_t loc_alpha = -13.8155f / T60; //-13.8155 == log(10^(-6))
		
		A[m_src*M_rcv + m_rcv] = finalPower / expf(loc_alpha*(w_center-tau_dp[m_src*M_rcv+m_rcv]));
		alpha[m_src*M_rcv + m_rcv] = loc_alpha;
		
		//printf("T60[%d][%d] = %f\n", m_src, m_rcv, T60);
		//printf("A[%d][%d] = %f\n", m_src, m_rcv, A[m_src*M_rcv + m_rcv]);
		//printf("alpha[%d][%d] = %f\n", m_src, m_rcv, alpha[m_src*M_rcv + m_rcv]);
	}
}

__global__ void diffRev_kernel(scalar_t* rir, scalar_t* tim, scalar_t* A, scalar_t* alpha, scalar_t* tau_dp, 
							   int M_src, int M_rcv, int nSamples) {
	
	int sample = blockIdx.x * blockDim.x + threadIdx.x;
	int m_src  = blockIdx.y * blockDim.y + threadIdx.y;
	int m_rcv  = blockIdx.z * blockDim.z + threadIdx.z;
	
	if (sample<nSamples && m_src<M_src && m_rcv<M_rcv) {
		// Get logistic distribution from uniform distribution
		scalar_t uniform = rir[m_src*M_rcv*nSamples + m_rcv*nSamples + sample];
		scalar_t logistic = 0.551329f * logf(uniform/(1.0f - uniform)); // 0.551329 == sqrt(3)/pi
		
		// Apply power envelope
		scalar_t pow_env = A[m_src*M_rcv+m_rcv] * expf(alpha[m_src*M_rcv+m_rcv] * (tim[sample]-tau_dp[m_src*M_rcv+m_rcv]));
		rir[m_src*M_rcv*nSamples + m_rcv*nSamples + sample] = sqrt(pow_env) * logistic;
		//if (sample==0) printf("A[%d][%d] = %f\n", m_src, m_rcv, A[m_src*M_rcv + m_rcv]);
	}
}

/***************************/
/* Auxiliar host functions */
/***************************/

scalar_t* cuda_rirGenerator(scalar_t* rir, scalar_t* x, scalar_t* amp, scalar_t* tau, int M, int N, int T, scalar_t Fs) {
	int initialReduction = initialReductionMin;
	while (M * T * ceil((float)N/initialReduction) > 1e9) initialReduction *= 2;
	
	int iniRIR_N = ceil((float)N/initialReduction);
	dim3 threadsPerBlockIni(nThreadsGen_t, nThreadsGen_m, nThreadsGen_n);
	dim3 numBlocksIni(ceil((float)T/threadsPerBlockIni.x), ceil((float)M/threadsPerBlockIni.y), iniRIR_N);
	
	scalar_t* initialRIR;
	gpuErrchk( hipMalloc(&initialRIR, M*T*iniRIR_N*sizeof(scalar_t)) );
	
	//printf("initialReduction = %d\n", initialReduction);
	//printf("generateRIR_kernel<<<(%d, %d, %d), (%d, %d, %d)>>>\n", numBlocksIni.x, numBlocksIni.y, numBlocksIni.z, threadsPerBlockIni.x, threadsPerBlockIni.y, threadsPerBlockIni.z);
	generateRIR_kernel<<<numBlocksIni, threadsPerBlockIni>>>( initialRIR, x, amp, tau, T, M, N, iniRIR_N, initialReduction, Fs );
	gpuErrchk( hipDeviceSynchronize() );
	gpuErrchk( hipPeekAtLastError() );
	
	dim3 threadsPerBlockRed(nThreadsRed, 1, 1);
	scalar_t* intermediateRIR;
	int intRIR_N;
	while (iniRIR_N > 2*nThreadsRed) {		
		intRIR_N = ceil((float)iniRIR_N / (2*nThreadsRed));
		gpuErrchk( hipMalloc(&intermediateRIR, intRIR_N * T * M * sizeof(scalar_t)) );

		dim3 numBlocksRed(intRIR_N, T, M);
		reduceRIR_kernel<<<numBlocksRed, threadsPerBlockRed, nThreadsRed*sizeof(scalar_t)>>>(
			initialRIR, intermediateRIR, M, T, iniRIR_N, intRIR_N);
		gpuErrchk( hipDeviceSynchronize() );
		gpuErrchk( hipPeekAtLastError() );
		
		gpuErrchk( hipFree(initialRIR) );
		initialRIR = intermediateRIR;		
		iniRIR_N = intRIR_N;
	}
	
	dim3 numBlocksEnd(1, T, M);
	reduceRIR_kernel<<<numBlocksEnd, threadsPerBlockRed, nThreadsRed*sizeof(scalar_t)>>>(
		initialRIR, rir, M, T, iniRIR_N, 1);
	gpuErrchk( hipDeviceSynchronize() );
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipFree(initialRIR) );
	
	return rir;
}

/**********************/
/* Principal function */
/**********************/

scalar_t* cuda_simulateRIR(scalar_t room_sz[3], scalar_t beta[6], scalar_t* h_pos_src, int M_src, scalar_t* h_pos_rcv, int M_rcv, 
						   int nb_img[3], scalar_t Tdiff, scalar_t Tmax, scalar_t Fs=16000.0f, scalar_t c=343.0f) {	
	// Copy host memory to GPU
	scalar_t *pos_src, *pos_rcv;
	gpuErrchk( hipMalloc(&pos_src, M_src*3*sizeof(scalar_t)) );
	gpuErrchk( hipMalloc(&pos_rcv, M_rcv*3*sizeof(scalar_t)) );
	gpuErrchk( hipMemcpy(pos_src, h_pos_src, M_src*3*sizeof(scalar_t), hipMemcpyHostToDevice ) );
	gpuErrchk( hipMemcpy(pos_rcv, h_pos_rcv, M_rcv*3*sizeof(scalar_t), hipMemcpyHostToDevice ) );
	
	
	// Use the ISM to calculate the amplitude and delay of each image
	dim3 threadsPerBlockISM(nThreadsISM_x, nThreadsISM_y, nThreadsISM_z);
	dim3 numBlocksISM(ceil((float)nb_img[0] / nThreadsISM_x), 
					  ceil((float)nb_img[1] / nThreadsISM_y), 
					  ceil((float)nb_img[2] / nThreadsISM_z));
	int shMemISM = (M_src + M_rcv) * 3 * sizeof(scalar_t);
	
	scalar_t* amp;
	gpuErrchk( hipMalloc(&amp, M_src*M_rcv*nb_img[0]*nb_img[1]*nb_img[2]*sizeof(scalar_t)) );
	scalar_t* tau;
	gpuErrchk( hipMalloc(&tau, M_src*M_rcv*nb_img[0]*nb_img[1]*nb_img[2]*sizeof(scalar_t)) );
	scalar_t* tau_dp; // Direct path delay
	gpuErrchk( hipMalloc(&tau_dp, M_src*M_rcv*sizeof(scalar_t)) );
	
	calcAmpTau_kernel<<<numBlocksISM, threadsPerBlockISM, shMemISM>>> (
		amp, tau, tau_dp,
		pos_src, pos_rcv, 
		room_sz[0], room_sz[1], room_sz[2], 
		beta[0], beta[1], beta[2], beta[3], beta[4], beta[5], 
		nb_img[0], nb_img[1], nb_img[2],
		M_src, M_rcv, c
	);
	gpuErrchk( hipDeviceSynchronize() );
	gpuErrchk( hipPeekAtLastError() );
	
	// Generate a vector with the time instant of each sample
	int nSamplesISM = ceil(Tdiff*Fs);
	int nSamples = ceil(Tmax*Fs);
	int nSamplesDiff = nSamples - nSamplesISM;
	//printf("nSamplesISM = %d\nnSamplesDiff = %d\nnSamples = %d\n", nSamplesISM, nSamplesDiff, nSamples);
	scalar_t* time;
	gpuErrchk( hipMalloc(&time, nSamples*sizeof(scalar_t)) );
	generateTime_kernel<<<ceil((float)nSamples/nThreadsTime), nThreadsTime>>>(time, Fs, nSamples);
	
	// Compute the RIRs as a sum of sincs
	int M = M_src * M_rcv;
	int N = nb_img[0] * nb_img[1] * nb_img[2];
	scalar_t* rirISM;
	gpuErrchk( hipMalloc(&rirISM, M*nSamplesISM*sizeof(scalar_t)) );
	cuda_rirGenerator(rirISM, time, amp, tau, M, N, nSamplesISM, Fs);
	
	// Compute the exponential power envelope parammeters of each RIR
	dim3 threadsPerBlockEnvPred(nThreadsEnvPred_x, nThreadsEnvPred_y, nThreadsEnvPred_z);
	dim3 numBlocksEnvPred(ceil((float)M_src / nThreadsEnvPred_x), 
						  ceil((float)M_rcv / nThreadsEnvPred_y), 1);
					  
	scalar_t* A; // pow_env = A * exp(alpha * (t-tau_dp))
	gpuErrchk( hipMalloc(&A, M_src*M_rcv*sizeof(scalar_t)) );
	scalar_t* alpha;
	gpuErrchk( hipMalloc(&alpha, M_src*M_rcv*sizeof(scalar_t)) );
	
	envPred_kernel<<<numBlocksEnvPred, threadsPerBlockEnvPred>>>(
			A, alpha, rirISM, tau_dp, M_src, M_rcv, nSamplesISM, Fs,
			room_sz[0], room_sz[1], room_sz[2], beta[0], beta[1], beta[2], beta[3], beta[4], beta[5]);
	gpuErrchk( hipDeviceSynchronize() );
	gpuErrchk( hipPeekAtLastError() );
	
	// Generate diffuse reverberation
	scalar_t* rirDiff; 
	gpuErrchk( hipMalloc(&rirDiff, M_src*M_rcv*nSamplesDiff*sizeof(scalar_t)) );
	
	if (nSamplesDiff != 0) {
		hiprandGenerator_t gen; // Fill rirDiff with random numbers with uniform distribution
		gpuErrchk( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
		gpuErrchk( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
		gpuErrchk( hiprandGenerateUniform(gen, rirDiff, M_src*M_rcv*nSamplesDiff) );
		gpuErrchk( hipDeviceSynchronize() );
		gpuErrchk( hipPeekAtLastError() );
		
		dim3 threadsPerBlockDiff(nThreadsDiff_t, nThreadsDiff_src, nThreadsDiff_rcv);
		dim3 numBlocksDiff(ceil((float)nSamplesDiff / nThreadsDiff_t),
							  ceil((float)M_src / nThreadsDiff_src), 
							  ceil((float)M_rcv / nThreadsDiff_rcv));
		diffRev_kernel<<<numBlocksDiff, threadsPerBlockDiff>>>(
				rirDiff, &time[nSamplesISM], A, alpha, tau_dp, M_src, M_rcv, nSamplesDiff);
		gpuErrchk( hipDeviceSynchronize() );
		gpuErrchk( hipPeekAtLastError() );
	}
	
	// Copy GPU memory to host
	int rirSizeISM = M_src * M_rcv * nSamplesISM * sizeof(scalar_t);
	int rirSizeDiff = M_src * M_rcv * nSamplesDiff * sizeof(scalar_t);
	scalar_t* h_rir = (scalar_t*) malloc(rirSizeISM+rirSizeDiff);
	
	hipPitchedPtr h_rir_pitchedPtr = make_hipPitchedPtr( (void*) h_rir, 
		(nSamplesISM+nSamplesDiff)*sizeof(scalar_t), nSamplesISM+nSamplesDiff, M_rcv );
	hipPitchedPtr rirISM_pitchedPtr = make_hipPitchedPtr( (void*) rirISM, 
		nSamplesISM*sizeof(scalar_t), nSamplesISM, M_rcv );
	hipPitchedPtr rirDiff_pitchedPtr = make_hipPitchedPtr( (void*) rirDiff, 
		nSamplesDiff*sizeof(scalar_t), nSamplesDiff, M_rcv );
	
	hipMemcpy3DParms parmsISM = {0};
	parmsISM.srcPtr = rirISM_pitchedPtr;
	parmsISM.dstPtr = h_rir_pitchedPtr;
	parmsISM.extent = make_hipExtent(nSamplesISM*sizeof(scalar_t), M_rcv, M_src);
	parmsISM.kind = hipMemcpyDeviceToHost;
	gpuErrchk( hipMemcpy3D(&parmsISM) );
	
	if (nSamplesDiff > 0) {
		hipMemcpy3DParms parmsDiff = {0};
		parmsDiff.srcPtr = rirDiff_pitchedPtr;
		parmsDiff.dstPtr = h_rir_pitchedPtr;
		parmsDiff.dstPos = make_hipPos(nSamplesISM*sizeof(scalar_t), 0, 0);
		parmsDiff.extent = make_hipExtent(nSamplesDiff*sizeof(scalar_t), M_rcv, M_src);
		parmsDiff.kind = hipMemcpyDeviceToHost;
		gpuErrchk( hipMemcpy3D(&parmsDiff) );
	}

	// Free memory
	gpuErrchk( hipFree(pos_src) );
	gpuErrchk( hipFree(pos_rcv) );
	gpuErrchk( hipFree(amp)	 );
	gpuErrchk( hipFree(tau)	 );
	gpuErrchk( hipFree(tau_dp)	 );
	gpuErrchk( hipFree(time)	 );
	gpuErrchk( hipFree(rirISM)	 );
	gpuErrchk( hipFree(A)		 );
	gpuErrchk( hipFree(alpha)	 );
	gpuErrchk( hipFree(rirDiff) );
	
	return h_rir;
}
